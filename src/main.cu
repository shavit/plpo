#include "hip/hip_runtime.h"
#include <cstdlib>
#include <stdio.h>
#include <math.h>

#include "../include/interpolate.cuh"
#include "../include/plpo.cuh"

extern "C" {
#include "../include/cli.h"
#include "../include/image.h"
}

__host__
int read_args(int argvc, char** argv, PLPO_CLIArgs_t* args) {
    if (argvc < 4) {
        return -1;
    }

    args->image_path = argv[1];
    args->filter_path = argv[2];
    args->out_image_path = argv[3];
    args->parallel = false;

    return 0;
}

void mem_assign_unflatten(unsigned char** output, unsigned char* input, const int w, const int h) {
    for (int i = 0; i < h; ++i) {
        for (int j = 0; j < w; ++j) {
            const int xy = i * w + j;
            output[i][j] = input[xy];
        }
    }
}

void mem_assign_flatten(unsigned char* output, unsigned char** input, const int w, const int h) {
    for (int i = 0; i < h; ++i) {
        for (int j = 0; j < w; ++j) {
            const int xy = i * w + j;
            output[xy] = input[i][j];
        }
    }
}

void checkCudaError(const char* msg) {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error: %s\n", msg);
        fprintf(stderr, "CUDA Error %d: %s\n", err, hipGetErrorString(err));

        exit(5);
    }
}

void determineCudaMaxBlockSize(int* numBlocks, int* maxBlockSize) {
    hipOccupancyMaxPotentialBlockSize(numBlocks, maxBlockSize, plpo_make_trilerp);
    //fprintf(stderr, "Determined cuda potential block size.\n\t%d blocks (numBlocks)\n\t%d grid size (maxBlockSize)\n", *numBlocks, *maxBlockSize);
}

__host__
int main(int argvc, char** argv) {
    int err = 0;

    PLPO_CLIArgs_t* args = (PLPO_CLIArgs_t*) malloc(sizeof(*args));
    if ((err = read_args(argvc, argv, args)) != 0) {
        return err;
    }

    PLPOImage_t* img = (PLPOImage_t*) malloc(sizeof(*img));
    plpo_image_init(img);
    if ((err = plpo_image_read(args->image_path, img)) != 0) {
        return err;
    }
    unsigned char* img_1d = (unsigned char*) malloc(sizeof(unsigned char) * img->row_length * img->height);
    mem_assign_flatten(img_1d, img->bytes, img->row_length, img->height);

    PLPOImage_t* lut = (PLPOImage_t*) malloc(sizeof(*lut));
    plpo_image_init(lut);
    if ((err = plpo_image_read(args->filter_path, lut)) != 0) {
        return err;
    }
    unsigned char* lut_1d = (unsigned char*) malloc(sizeof(unsigned char) * lut->row_length * lut->height);
    mem_assign_flatten(lut_1d, lut->bytes, lut->row_length, lut->height);

    int numBlocks;
    int maxBlockSize;
    determineCudaMaxBlockSize(&numBlocks, &maxBlockSize);
    const dim3 dim_block(maxBlockSize / 32 / 2, maxBlockSize / 32 / 2, 1);
    dim3 dim_grid((img->height - 1) / dim_block.x + 1, (img->width - 1) / dim_block.y + 1, 1); // round up

    unsigned char* cuimg_mat;
    hipMalloc((void**) &cuimg_mat, sizeof(unsigned char) * img->row_length * img->height);
    checkCudaError("After allocating img");
    hipMemcpy(cuimg_mat, img_1d, sizeof(unsigned char) * img->row_length * img->height, hipMemcpyHostToDevice);
    checkCudaError("After copying img row to device");

    unsigned char* culut_mat;
    hipMalloc((void**) &culut_mat, sizeof(unsigned char) * lut->row_length * lut->height);
    checkCudaError("After allocating lut");
    hipMemcpy(culut_mat, lut_1d, sizeof(unsigned char) * lut->row_length * lut->height, hipMemcpyHostToDevice);
    checkCudaError("After copying lut row to device");

    const int m = floor(cbrt(lut->width));
    const int m2 = m * m; // block range 0..m2
    const float sig = (m2 - 1) / (float)0xff;
    plpo_make_trilerp<<<dim_grid, dim_block>>>(culut_mat, cuimg_mat, img->width, img->height, m, m2, sig, 3);
    checkCudaError("Error returning from kernel");

    hipMemcpy(img_1d, cuimg_mat, sizeof(unsigned char) * img->row_length * img->height, hipMemcpyDeviceToHost);
    checkCudaError("Error after data copied to host");
    mem_assign_unflatten(img->bytes, img_1d, img->row_length, img->height);

    hipFree(cuimg_mat);
    hipFree(culut_mat);
    free(img_1d);
    free(lut_1d);

    if ((err = plpo_image_write(args->out_image_path, img)) != 0) {
        plpo_image_destroy(img);
        plpo_image_destroy(lut);
        return err;
    }

    plpo_image_destroy(img);
    plpo_image_destroy(lut);

    fprintf(stderr, "Saved %s\n", args->out_image_path);

    return 0;
}    
